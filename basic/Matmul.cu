
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

using namespace std;

#define BLOCK_SIZE 16

__global__ void multiplication(float *A, float *B, float *C, int N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0.f;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
    // Initialize clock variables
    clock_t cpuClock;
    clock_t gpuClock;

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	int N,K;
	K = 100;			
	N = K*BLOCK_SIZE;
	
	cout << "Matrix size: " << N << "x" << N << endl << endl;

	// Allocate memory on the host
	float *hA,*hB,*hC;
	hA = new float[N*N];
	hB = new float[N*N];
	hC = new float[N*N];

	// Initialize matrices on the host
	for (int j=0; j<N; j++){
	    for (int i=0; i<N; i++){
	    	hA[j*N+i] = 2.f*(j+i);
			hB[j*N+i] = 1.f*(j-i);
	    }
	}

	// Allocate memory on the device
	int size = N*N*sizeof(float);	// Size of the memory in bytes
	float *dA,*dB,*dC;
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);

	dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(K,K);
	
	// Copy matrices from the host to device
	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
	
	// Do the matrix multiplication on the GPU
	gpuClock = clock();
	multiplication<<<grid,threadBlock>>>(dA,dB,dC,N);
	gpuClock = clock() - gpuClock;
	
	
	// Now do the matrix multiplication on the CPU
	cpuClock = clock();
	float sum;
	for (int row=0; row<N; row++){
		for (int col=0; col<N; col++){
			sum = 0.f;
			for (int n=0; n<N; n++){
				sum += hA[row*N+n]*hB[n*N+col];
			}
			hC[row*N+col] = sum;
		}
	}
	cpuClock = clock() - cpuClock;
	
	// Allocate memory to store the GPU answer on the host
	float *C;
	C = new float[N*N];
	
	// Now copy the GPU result back to CPU
	hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);
	
	// Check the result and make sure it is correct
	for (int row=0; row<N; row++){
		for (int col=0; col<N; col++){
			if( C[row*N+col] != hC[row*N+col] ){
				cout << "Wrong answer!" << endl;
				row = col = N;
			}
		}
	}
	
	printf("The CPU took %f seconds to perform matrix multiplication. \n", ((float)cpuClock)/CLOCKS_PER_SEC);
	printf("The GPU took %f seconds to perform matrix multiplication. \n", ((float)gpuClock)/CLOCKS_PER_SEC);
		
}
